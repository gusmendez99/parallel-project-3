#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 To build use  : make global
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;


//************************************************************************
// Check return function 
//************************************************************************

#define CUDA_CHECK_RETURN(value)                                    \
{                                                                   \
  hipError_t _m_cudaStat = value;                                  \
  if (_m_cudaStat != hipSuccess)                                   \
  {                                                                 \
    fprintf(stderr, "Error %s at line %d in file %s\n",             \
      hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
    exit(1);                                                        \
  }                                                                 \
}



//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}


// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  // Calculo global ID
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  // TODO: Explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  /* R// Para calcular xCoord se debe realizar el módulo de gloID con el ancho de la imagen
   con el fin de poder ubicar en qué posición en el eje x se encuentra nuestro hilo.
   Luego, se realiza un corrimiento respecto a centro de la imagen.
   Para la coordenada en y se debe divide el id de nuestro hilo por el ancho
   esto nos ubica en el eje vertical. Luego este valor es restado respecto al centro de la image
   y con esto se obtiene las coordenadas a partir del id global de nuestro hilo
   que son relativas a nuestro plano

  */
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;
      // TODO: Debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
      // R// Se debe utilizar atomic ya que estamos modificando el arreglo global, el cual es compartido
      // por todos los hilos. De esta manera garantizamos que la transacción se realice adecuadamente y no
      // ocasione race conditions.
      atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
    }
  }
}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);
  hipEvent_t start, stop;
  float time;

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpy(d_Cos, pcCos, sizeof (float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof (float) * degreeBins, hipMemcpyHostToDevice);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);

  //Get time with events
  CUDA_CHECK_RETURN( hipEventCreate(&start) );
  CUDA_CHECK_RETURN( hipEventCreate(&stop) );
  CUDA_CHECK_RETURN( hipEventRecord(start, 0) );

  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  CUDA_CHECK_RETURN( hipEventRecord(stop, 0) );
  CUDA_CHECK_RETURN( hipEventSynchronize(stop) );
  CUDA_CHECK_RETURN( hipEventElapsedTime(&time, start, stop) );

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");
  printf("EXEC TIME:  %3.1f ms \n", time);

  // Clean-up
  hipFree ((void *) d_Cos);
  hipFree ((void *) d_Sin);
  hipFree ((void *) d_in);
  hipFree ((void *) d_hough);
  free (h_hough);
  free (cpuht);
  free (pcCos);
  free (pcSin);
  hipDeviceReset ();

  return 0;
}
