#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas, Gus Mendez & Roberto Figueroa
 To build use  : make constant
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//************************************************************************
// Check return function 
//************************************************************************

#define CUDA_CHECK_RETURN(value)                                    \
{                                                                   \
  hipError_t _m_cudaStat = value;                                  \
  if (_m_cudaStat != hipSuccess)                                   \
  {                                                                 \
    fprintf(stderr, "Error %s at line %d in file %s\n",             \
      hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
    exit(1);                                                        \
  }                                                                 \
}


//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// Usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];


// Kernel memoria Constante
__global__ void GPU_HoughTranShared(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  // Calculo global ID
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block
  
  int i;
  int locID = threadIdx.x;
  int xCent = w / 2;
  int yCent = h / 2;

  // TODO: Explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  // R// xyz
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  // Use shared memory here for acc variable
  __shared__ int localAcc[degreeBins * rBins];
  // Initialize
  for (i = locID; i < degreeBins * rBins; i += blockDim.x)
    localAcc[i] = 0;

  // warps sync
  __syncthreads ();

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;

      // TODO: Debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
      // R// xyz
      atomicAdd (localAcc + (rIdx * degreeBins + tIdx), 1);
    }
  }

  // warps sync again
  __syncthreads ();

  // atomic op, add local acc to the global memory acc
  for (i = locID ; i < degreeBins * rBins ; i += blockDim.x)
    atomicAdd (acc + i, localAcc[i]);

}


//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);
  hipEvent_t start, stop;
  float time;

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos(rad);
    pcSin[i] = sin(rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // Eventualmente volver memoria global
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);
  //Get time with events
  CUDA_CHECK_RETURN( hipEventCreate(&start) );
  CUDA_CHECK_RETURN( hipEventCreate(&stop) );
  CUDA_CHECK_RETURN( hipEventRecord(start, 0) );

  // NOTE: We're not passing d_Sin & d_Cos to the kernel, it's constant memory!
  GPU_HoughTranShared <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

  CUDA_CHECK_RETURN( hipEventRecord(stop, 0) );
  CUDA_CHECK_RETURN( hipEventSynchronize(stop) );
  CUDA_CHECK_RETURN( hipEventElapsedTime(&time, start, stop) );

  hipDeviceSynchronize();

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");
  printf("EXEC TIME:  %3.1f ms \n", time);

  // Clean-up
  hipFree ((void *) d_Cos);
  hipFree ((void *) d_Sin);
  hipFree ((void *) d_in);
  hipFree ((void *) d_hough);
  free (h_hough);
  free (cpuht);
  free (pcCos);
  free (pcSin);
  hipDeviceReset ();

  return 0;
}
